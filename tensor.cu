#include <cstring>
#include <stdlib.h>

#include "tensor.h"
#include "util.h"
using namespace std;

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

Tensor::Tensor(const vector<int> &shape_) {
  reshape(shape_);
  // buf = (float *)malloc(n * sizeof(float));
  CHECK_CUDA(hipMalloc(&gbuf, n * sizeof(float)));
}

Tensor::Tensor(float *data, const vector<int> &shape_) {
  reshape(shape_);
  // buf = (float *)malloc(n * sizeof(float));
  CHECK_CUDA(hipMalloc(&gbuf, n * sizeof(float)));
  
  // memcpy(buf, data, get_elem() * sizeof(float));
  hipMemcpy(gbuf, data, get_elem() * sizeof(float), hipMemcpyHostToDevice);
}

Tensor::~Tensor() {
  //free(buf); 
  hipFree(gbuf);
}

void Tensor::load(const char *filename) {
  size_t m;
  float *tmp;
  tmp = (float *)read_binary(filename, &m);
  CHECK_CUDA(hipMemcpy(gbuf, tmp, sizeof(tmp), hipMemcpyHostToDevice));
  n = m;
  reshape({n});
}
void Tensor::save(const char *filename) { 
    float *tmp;
    CHECK_CUDA(hipMemcpy(tmp, gbuf, get_elem() * sizeof(float), hipMemcpyDeviceToHost));
    write_binary(tmp, filename, n);
  }

int Tensor::get_elem() { return n; }

void Tensor::reshape(const vector<int> &shape_) {
  n = 1;
  ndim = shape_.size(); // ndim<=4
  for (int i = 0; i < ndim; i++) {
    shape[i] = shape_[i];
    n *= shape[i];
  }
}
